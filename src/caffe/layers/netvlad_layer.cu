#include "hip/hip_runtime.h"
/* UNC Software �Learned Contextual Feature Reweighting for Image Geo-Localization�
Copyright (C) 2017 The University of North Carolina at Chapel Hill
All rights reserved.
Written by Hyo Jin Kim (hyojin@cs.unc.edu)
*/

#include <vector>

#include <algorithm>
#include <cmath>
#include <cfloat>

#include "caffe/layers/netvlad_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

	
template <typename Dtype>
__global__ void NetvladTranspose(const int n, const int d_local, const int d_area, Dtype* trans_mat, Dtype* orig_mat) {
		CUDA_KERNEL_LOOP(index, n) {
			const int p = index / d_local;
			const int ho = index % d_local;
			const int res_idx = ho * d_area + p;
			trans_mat[index] = orig_mat[res_idx];
			//const int label_value = static_cast<int>(label[n * spatial_dim + s]);
			//loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
		}
}


template <typename Dtype>
void NetvladLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

	// Output (NetVlad)
	Dtype* top_data = top[0]->mutable_gpu_data();
	caffe_gpu_set(top[0]->count(), Dtype(0), top_data);

	// Offsets (Cluster Center)
	const Dtype* clst_center = this->blobs_[0]->gpu_data();
	const Dtype* soft_assign = bottom[0]->gpu_data(); // *Note: cpu data
	const Dtype* bottom_data = bottom[1]->gpu_data();

	const int num = bottom[1]->num(); // batch size :shape(0)
	const int width = bottom[1]->width(); // batch size :shape(3)
	const int height = bottom[1]->height(); // batch size :shape(2)
	// int channels = bottom[1]->channels();
	const int area = width * height;

	const int dim = bottom[1]->count() / bottom[1]->num();  // width*height*channel (contains res of one image)

	NetvladParameter netvlad_param = this->layer_param_.netvlad_param();
	const int cluster_num = netvlad_param.cluster_num();      // number of clusters K
	const int local_dim = netvlad_param.local_dim();  // local feature dim D (e.g. 128 for SIFT), output of previous layer's depth
	const int vlad_dim = netvlad_param.vlad_dim(); // vlad dim, which should match D*K

	Dtype* residual_data = residual_.mutable_gpu_data();
	Dtype* tresidual_data = tresidual_.mutable_gpu_data();
	
	caffe_gpu_set(residual_.count(), Dtype(0), residual_data);
	caffe_gpu_set(tresidual_.count(), Dtype(0), tresidual_data);

	if (residual_.count() != dim || tresidual_.count() != dim){
		LOG(FATAL) << "temporal storage residual or tresidual does not match size of C*W*H (dim).";
	}

	for (int n = 0; n < num; ++n) {
		// for every image,
		for (int k = 0; k < cluster_num; ++k){
			// for every cluster,
			Dtype* k_cum_residue = top_data + n * vlad_dim + k * local_dim;
			caffe_gpu_memcpy(sizeof(Dtype)* dim, bottom_data + n * dim, residual_data);
			const Dtype* curr_center = clst_center + k * local_dim;

			// residual_data = feature - cluster center 
			caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, local_dim, area, 1, (Dtype)1., curr_center, sum_multiplier_.gpu_data(), (Dtype)1., residual_data); //(512x(13x13))
			// assignment to this cluster 
			const Dtype* curr_assignment = soft_assign + n * (cluster_num * area) + k * area; // soft_assign: (100x64x13x13)

			/*
			NetvladTranspose<Dtype> << <CAFFE_GET_BLOCKS(mat_dim), CAFFE_CUDA_NUM_THREADS >> >(
				mat_dim, local_dim, area, tresidual_data, residual_data);
			CUDA_POST_KERNEL_CHECK;
			*/
			
			caffe_gpu_gemv(CblasNoTrans, local_dim, area, (Dtype)1., residual_data, curr_assignment, (Dtype)0., k_cum_residue);

		}
		
	}

}

template <typename Dtype>
void NetvladLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
   
	//Backward_cpu(top, propagate_down, bottom);

	const Dtype* top_diff = top[0]->gpu_diff();
	// assign (a) : after softmax
	const Dtype* assign_data = bottom[0]->gpu_data();
	Dtype* assign_diff = bottom[0]->mutable_gpu_diff();
	// x data
	const Dtype* x_data = bottom[1]->gpu_data();
	Dtype* x_diff = bottom[1]->mutable_gpu_diff();
	// offset
	const Dtype* clst_center = this->blobs_[0]->gpu_data();
	Dtype* clst_center_diff = this->blobs_[0]->mutable_gpu_diff(); // offset diff

	const int num = bottom[1]->num(); // batch size :shape(0)
	const int width = bottom[1]->width(); // batch size :shape(3)
	const int height = bottom[1]->height(); // batch size :shape(2)
	// int channels = bottom[1]->channels();
	const int area = width * height;
	const int dim = bottom[1]->count() / bottom[1]->num();  // width*height*channel (contains res of one image)

	NetvladParameter netvlad_param = this->layer_param_.netvlad_param();
	const int cluster_num = netvlad_param.cluster_num();      // number of clusters K
	const int local_dim = netvlad_param.local_dim();  // local feature dim D (e.g. 128 for SIFT), output of previous layer's depth
	const int vlad_dim = netvlad_param.vlad_dim(); // vlad dim, which should match D*K

	Dtype* residual_data = residual_.mutable_gpu_data();
	caffe_gpu_set(residual_.count(), Dtype(0.0), residual_data); // for the safety

	caffe_gpu_set(vlad_dim, Dtype(0.0), clst_center_diff);

	Dtype sum_ak;

	for (int n = 0; n < num; ++n) {
		// for each batch
		const Dtype* x_mat = x_data + n* (local_dim * area);
		Dtype* x_diff_mat = x_diff + n* (local_dim * area);
		const Dtype* assign_mat = assign_data + n * (cluster_num * area);
		Dtype* assign_diff_mat = assign_diff + n * (cluster_num * area);
		const Dtype* top_diff_mat = top_diff + n * vlad_dim;

		// dz/da
		for (int k = 0; k < cluster_num; ++k){
			caffe_copy(dim, x_mat, residual_data); // (512x13x13)

			const Dtype* curr_center = clst_center + k * local_dim;
			const Dtype* top_diff_k = top_diff_mat + k * local_dim;  // Good. 
			Dtype* assign_diff_k = assign_diff_mat + k * area;
			const Dtype* assign_mat_k = assign_mat + k * area;

			// subtract cluster center
			caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, local_dim, area, 1, (Dtype)1., curr_center, sum_multiplier_.gpu_data(), (Dtype)1., residual_data); //(512x(13x13))

			// sum wrt #local_dim
			caffe_gpu_gemm<Dtype>(CblasTrans, CblasTrans, area, 1, local_dim, (Dtype)1., residual_data, top_diff_k, (Dtype)0., assign_diff_k); //((13x13)x1)
			
			// dz/dc
			Dtype* clst_center_diff_k = clst_center_diff + k * local_dim;
			
			caffe_gpu_dot<Dtype>(area, assign_mat_k, sum_multiplier_.gpu_data(), &sum_ak);

			caffe_gpu_axpy<Dtype>(local_dim, sum_ak, top_diff_k, clst_center_diff_k); // 1x 512

		}

		// dz/dx
		caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, local_dim, area, cluster_num, (Dtype)1., top_diff_mat, assign_mat, (Dtype)0., x_diff_mat);
	}

}

INSTANTIATE_LAYER_GPU_FUNCS(NetvladLayer);

}  // namespace caffe

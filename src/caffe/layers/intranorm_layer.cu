#include "hip/hip_runtime.h"
/* UNC Software �Learned Contextual Feature Reweighting for Image Geo-Localization�
Copyright (C) 2017 The University of North Carolina at Chapel Hill
All rights reserved.
Written by Hyo Jin Kim (hyojin@cs.unc.edu)
*/
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/intranorm_layer.hpp"

namespace caffe {

template <typename Dtype>
void IntranormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

	IntranormParameter intranorm_param = this->layer_param_.intranorm_param();
	int cluster_num = intranorm_param.cluster_num();
	Dtype eps_ = Dtype(0.000000000001);
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype normsqr;
  int n = bottom[0]->num();
  int d = bottom[0]->count() / n;

  int subdim = d / cluster_num;

  caffe_gpu_powx(n*d, bottom_data, Dtype(2), squared_data);
  for (int i=0; i<n; ++i) {
	  Dtype* curr_sqr_data = squared_data + i*d;
	  const Dtype* curr_bottom_data = bottom_data + i*d;
	  Dtype* curr_top_data = top_data + i*d;
	  for (int k = 0; k < cluster_num; ++k){
		  caffe_gpu_asum<Dtype>(subdim, curr_sqr_data + k*subdim, &normsqr);
		  caffe_gpu_scale<Dtype>(subdim, pow(normsqr + eps_, -0.5), curr_bottom_data + k*subdim, curr_top_data + k*subdim);
	  }
  }
}

template <typename Dtype>
void IntranormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

	IntranormParameter intranorm_param = this->layer_param_.intranorm_param();
	int cluster_num = intranorm_param.cluster_num();

  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int n = top[0]->num();
  int d = top[0]->count() / n;

  int subdim = d / cluster_num;

  Dtype a;
  Dtype eps_ = Dtype(0.000000000001);
  for (int i=0; i<n; ++i) {
	  const Dtype* curr_top_data = top_data + i*d;
	  const Dtype* curr_top_diff = top_diff + i*d;
	  const Dtype* curr_bottom_data = bottom_data + i*d;
	  Dtype* curr_bottom_diff = bottom_diff + i*d;
	  for (int k = 0; k < cluster_num; ++k){
		  caffe_gpu_dot(subdim, curr_top_data + k*subdim, curr_top_diff + k*subdim, &a);
		  caffe_gpu_scale(subdim, a, curr_top_data + k*subdim, curr_bottom_diff + k*subdim);
		  caffe_gpu_sub(subdim, curr_top_diff + k*subdim, curr_bottom_diff + k*subdim, curr_bottom_diff + k*subdim);
		  caffe_gpu_dot(subdim, curr_bottom_data + k*subdim, curr_bottom_data + k*subdim, &a);
		  caffe_gpu_scale(subdim, Dtype(pow(a + eps_, -0.5)), curr_bottom_diff + k*subdim, curr_bottom_diff + k*subdim);
	  }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(IntranormLayer);


}  // namespace caffe

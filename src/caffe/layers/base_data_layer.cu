#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void ReidPrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  ReidBatch<Dtype>* batch = this->prefetch_full_.pop("Data layer prefetch queue empty");
  // CHECK
  CHECK_EQ(top[0]->count(), batch->data_.count()*2);
  // Reshape to loaded data.
  top[0]->Reshape(batch->data_.num()*2, batch->data_.channels(), batch->data_.height(), batch->data_.width());
  // Copy the data
  caffe_copy(batch->data_.count(),  batch->data_.gpu_data(),  top[0]->mutable_gpu_data());
  caffe_copy(batch->datap_.count(), batch->datap_.gpu_data(), top[0]->mutable_gpu_data()+batch->data_.count());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    vector<int> shape = batch->label_.shape();
    CHECK_LT(shape.size(), 2);
    CHECK_EQ(top[1]->count(), batch->label_.count()*2);
    shape[0] *= 2;
    top[1]->Reshape(shape);
    // Copy the labels.
    caffe_copy(batch->label_.count(),  batch->label_.gpu_data(),  top[1]->mutable_gpu_data());
    caffe_copy(batch->labelp_.count(), batch->labelp_.gpu_data(), top[1]->mutable_gpu_data()+batch->label_.count());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void MsPrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  MsBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    for (int nn = 0; nn < batch->labels_.size(); nn++) {
      // Reshape to loaded labels.
      top[nn+1]->ReshapeLike(*batch->labels_[nn]);
      // Copy the labels.
      caffe_copy(batch->labels_[nn]->count(), batch->labels_[nn]->gpu_data(),
                 top[nn+1]->mutable_gpu_data());
    }
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}


INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(ReidPrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(MsPrefetchingDataLayer);

}  // namespace caffe
